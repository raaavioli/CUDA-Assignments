#include "hip/hip_runtime.h"
#include <stdio.h>
#include <chrono>

#define PI 3.14159265358
#define NUM_PARTICLES 10000
#define NUM_ITERATIONS 1000

struct Particle {
    float3 position;
    float3 velocity;
};

__host__
__device__
float3 operator+ (float3 a, float3 b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__
void cu_move_particles(Particle* particles, int num_particles, int iteration, int num_iterations) 
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= num_particles) return;

    float angle = PI * 2.0f * (id / (float) num_particles) * (iteration / (float) num_iterations);
    float3 dv = make_float3(sin(angle), cos(angle), sin(angle));
    particles[id].velocity = particles[id].velocity + dv;
    particles[id].position = particles[id].position + particles[id].velocity;
}

__host__
void move_particles(Particle* particles, int num_particles, int iteration, int num_iterations)
{
    for (int i = 0; i < num_particles; i++)
    {
        float angle = PI * 2.0f * (i / (float) num_particles) * (iteration / (float) num_iterations);
        float3 dv = make_float3(sin(angle), cos(angle), sin(angle));
        particles[i].velocity = particles[i].velocity + dv;
        particles[i].position = particles[i].position + particles[i].velocity;
    }
}

int main(int argc, char** argv)
{
    using namespace std::chrono;

    int num_particles = NUM_PARTICLES;
    int num_iterations = NUM_ITERATIONS;
    if (argc == 3) {
        num_particles = atoi(argv[1]);
        num_iterations = atoi(argv[2]);
        printf("Running simulation...\nnum particles: %d\nnum iterations: %d\n", num_particles, num_iterations);
    } else {
        printf("./bin <num_particles> <num_iterations>\n");
        return;
    }

    int buffer_size = sizeof(Particle) * num_particles;

    Particle* particles = (Particle*) malloc(buffer_size);
    for (int i = 0; i < num_particles; i++)
    {
        particles[i].velocity = {0};
        particles[i].position = {0};
    }

    // CPU Simulation
    printf("Simulating particles on CPU… ");
    high_resolution_clock::time_point t_cpu_start = high_resolution_clock::now();
    for (int i = 0; i < num_iterations; i++) 
    {
        move_particles(particles, num_particles, i, num_iterations);
    }
    high_resolution_clock::time_point t_cpu_end = high_resolution_clock::now();
    printf("Done!\n");

    // GPU Simulation
    printf("Simulating particles on GPU… block-size: ");
    Particle* cu_particles = 0;
    hipMalloc(&cu_particles, buffer_size);
    Particle* cu_particles_res = (Particle*) malloc(buffer_size);
    const int block_iterations = 5;
    duration<double> cu_durations_per_blocksize[block_iterations];
    duration<double> cu_mem_durations_per_blocksize[block_iterations];
    int block_size = 16;
    for (int i = 0; i < 5; i++) {
        printf("%d..., ", block_size);
        high_resolution_clock::time_point t_cu_mem_start = high_resolution_clock::now();
        hipMemset(cu_particles, 0, buffer_size);
        int num_blocks = (num_particles + block_size - 1) / block_size;
        high_resolution_clock::time_point t_cu_start = high_resolution_clock::now();
        for (int j = 0; j < num_iterations; j++) 
        {
            cu_move_particles<<<num_blocks, block_size>>>(cu_particles, num_particles, j, num_iterations);
            // hipDeviceSynchronize should not be needed here as each thread within a block operates only
            // on the same memory region, and will continue to the next once finished with previous iteration
        }
        hipDeviceSynchronize();
        high_resolution_clock::time_point t_cu_end = high_resolution_clock::now();
        cu_durations_per_blocksize[i] = duration_cast<duration<double>>(t_cu_end - t_cu_start);
        block_size *= 2;

        hipMemcpy(cu_particles_res, cu_particles, buffer_size, hipMemcpyDeviceToHost);
        high_resolution_clock::time_point t_cu_mem_end = high_resolution_clock::now();
        cu_mem_durations_per_blocksize[i] = duration_cast<duration<double>>(t_cu_mem_end - t_cu_mem_start);
        hipDeviceSynchronize();
    }
    printf("Done!!\n");

    // Equality comparison
    bool equal = true;
    printf("Comparing the output for each implementation… ");
    for (int i = 0; i < num_particles; i++)
    {
        // Divide difference by the minimum magnitude to get a fair comparison for large and 
        // small values having some rounding error in the least significant bits
        float minx = min(cu_particles_res[i].position.x, particles[i].position.x);
        float miny = min(cu_particles_res[i].position.y, particles[i].position.y);
        float minz = min(cu_particles_res[i].position.z, particles[i].position.z);
        if (abs((cu_particles_res[i].position.x - particles[i].position.x) / minx) > 1e-6 &&
            abs((cu_particles_res[i].position.y - particles[i].position.y) / miny) > 1e-6 && 
            abs((cu_particles_res[i].position.z - particles[i].position.z) / minz) > 1e-6 ) 
        {
            equal = false;
            printf("\ngpu: {%f, %f, %f} \ncpu: {%f, %f, %f}\n",    
                cu_particles_res[i].position.x, cu_particles_res[i].position.y, cu_particles_res[i].position.z, 
                particles[i].position.x, particles[i].position.y, particles[i].position.z);
            break;
        }
    }
    if (equal)
        printf("Correct!\n");
    else 
        printf("!!!!! Incorrect !!!!!\n");


    // Timing comparison
    duration<double> cpu_time = duration_cast<duration<double>>(t_cpu_end - t_cpu_start);
    
    printf("cpu (ms)\tcu16 (ms)\tcu32 (ms)\tcu64 (ms)\tcu128 (ms)\tcu256 (ms)\n");
    printf("%f\t", cpu_time.count() * 1e3);
    for (int i = 0; i < 5; i++) {
        //printf("%f\t%f\t", cu_durations_per_blocksize[i].count() * 1e3, cu_mem_durations_per_blocksize[i].count() * 1e3);
        printf("%f\t", cu_durations_per_blocksize[i].count() * 1e3);
    }
    printf("\n");

    hipFree(cu_particles);

    return 0;
}